
#include <hip/hip_runtime.h>
#include <stdio.h>
# define N 500

__global__
void sum(float *array, float *sum){
    for(int i=0; i<N; i++){
        *sum += *(array + i);
    }
}

int main(){
    float *array, *summation, avg;
    float *a, summation_host = 0.0f;
    int size = sizeof(float);
    for(int i=0; i < N; i++){
        *(a + i) = i * 2.0f;
    }
    hipMalloc(&array, N*size);
    hipMalloc(&summation, size);

    hipMemcpy(array, &a, N*size, hipMemcpyHostToDevice);
    hipMemcpy(summation, &summation_host, size, hipMemcpyHostToDevice);

    sum<<<1, 10>>>(array, summation);

    hipDeviceSynchronize();

    hipMemcpy(&summation_host, summation, sizeof(float), hipMemcpyDeviceToHost);
    avg = summation_host / N;

    hipFree(array);
    hipFree(summation);
    printf("avg is: %f", avg);
    return 0;
}